#include "hip/hip_runtime.h"
// Thread-based parallelism on GPU

// Each thread computes a part of the shortest path matrix.

#include <stdio.h>
#include <hip/hip_runtime.h>

#define INF 99999
#define V 4  // Adjust as needed

__global__ void floydWarshallKernel( int *dist, int k, int V ){
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    int ij = i * V + j;
    int ik = i * V + k;
    int kj = k * V + j;

    if( dist[ik] != INF && dist[kj] != INF && dist[ik] + dist[kj] < dist[ij] ){
        dist[ij] = dist[ik] + dist[kj];
    }
}

void floydWarshall( int graph[V][V] ){
    int *d_dist;
    size_t size = V * V * sizeof( int );

    hipMalloc( &d_dist, size );
    hipMemcpy( d_dist, graph, size, hipMemcpyHostToDevice );

    for( int k = 0 ; k < V ; k++ ){
        floydWarshallKernel<<<V, V>>>( d_dist, k, V );
        hipDeviceSynchronize();
    }

    hipMemcpy( graph, d_dist, size, hipMemcpyDeviceToHost );
    hipFree( d_dist );
}

void printSolution( int dist[V][V] ){
    for( int i = 0 ; i < V ; i++ ){
        for( int j = 0 ; j < V ; j++ ){
            if( dist[i][j] == INF ) printf( "INF " );
            else printf( "%d ", dist[i][j] );
        }
        printf( "\n" );
    }
}

int main(){
    int graph[V][V] = {
        { 0, 3, INF, 5 },
        { 2, 0, INF, 4 },
        { INF, 1, 0, INF },
        { INF, INF, 2, 0 }
    };

    floydWarshall( graph );
    printSolution( graph );

    return 0;
}

// Each thread computes a part of the shortest path update.
// Uses grid-stride looping to allow parallel execution.
// Synchronizes after each k iteration to ensure consistency.
